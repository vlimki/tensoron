
#include <hip/hip_runtime.h>
extern "C" __global__ void vec_add(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}
